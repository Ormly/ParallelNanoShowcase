#include "hip/hip_runtime.h"
#include "kernel.h"


__global__ void matrixMultiplication(const int *a, const int *b, int *c, int matrixDim) 
{
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Iterate over row, and down column
  c[row * N + col] = 0;
  for (int k = 0; k < N; k++) {
    // Accumulate results for a single element
    c[row * N + col] += a[row * N + k] * b[k * N + col];
  }
}

void launchKernel(const int THREADS, const int BLOCKS, const int *a, const int *b, int *c, int matrixDim)
{
	
	
	// Use dim3 structs for block  and grid dimensions
	dim3 threads(THREADS, THREADS);
	dim3 blocks(BLOCKS, BLOCKS);
	
	//Launch Kernel
	matrixMultiplication<<<blocks, threads>>>(a, b, c, N);
}
